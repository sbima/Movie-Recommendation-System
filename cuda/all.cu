#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h> 
#include <hipsolver.h>

void printMatrix(int m, int n, const double*A, int total_rows, const char* name) 
{ 
	for(int row = 0 ; row < m ; row++)
	{ 
		for(int col = 0 ; col < n ; col++)
		{ 
			double Areg = A[row + col*total_rows]; 
			printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg); 
		} 
	} 
} 

void lin_alg_solve(double * XC, double A[9], double B[3], const int m, const int nrhs)
{
	hipsolverHandle_t cusolverH = NULL;
	hipblasHandle_t cublasH = NULL; 
	hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS; 
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS; 
	hipError_t cudaStat1 = hipSuccess; 
	hipError_t cudaStat2 = hipSuccess; 
	hipError_t cudaStat3 = hipSuccess; 
	hipError_t cudaStat4 = hipSuccess; 
	//const int m = 3; 
	const int lda = m; 
	const int ldb = m; 
	//const int nrhs = 1;
	
	//Create the library handle and load the data (starts here)
	
	//double A[lda*m] = { 21.81678168,  15.31087255,  26.18776594, 15.31087255,  13.93152484,  23.92113599, 26.18776594,  23.92113599,  41.50060023};
	
	//double B[ldb*nrhs] = { 15.94772944,  16.57202022,  28.49909096}; 
	//double XC[ldb*nrhs]; // solution matrix from GPU 
	double *d_A = NULL; // linear memory of GPU 
	double *d_tau = NULL; // linear memory of GPU 
	double *d_B = NULL; 
	int *devInfo = NULL; // info in gpu (device copy) 
	double *d_work = NULL; 
	int lwork = 0; 
	
	int info_gpu = 0; 
	
	const double one = 1;
	
	printf("A = (matlab base-1)\n"); 
	printMatrix(m, m, A, lda, "A"); 
	printf("=====\n"); 
	printf("B = (matlab base-1)\n"); 
	printMatrix(m, nrhs, B, ldb, "B"); 
	printf("=====\n");
	
	// step 1: create cusolver/cublas handle
	cusolver_status = hipsolverDnCreate(&cusolverH); 
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
	
	cublas_status = hipblasCreate(&cublasH); 
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
	
	// step 2: copy A and B to device
	cudaStat1 = hipMalloc ((void**)&d_A , sizeof(double) * lda * m); 
	cudaStat2 = hipMalloc ((void**)&d_tau, sizeof(double) * m); 
	cudaStat3 = hipMalloc ((void**)&d_B , sizeof(double) * ldb * nrhs); 
	cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int)); 
	assert(hipSuccess == cudaStat1); 
	assert(hipSuccess == cudaStat2); 
	assert(hipSuccess == cudaStat3); 
	assert(hipSuccess == cudaStat4); 
	
	cudaStat1 = hipMemcpy(d_A, A, sizeof(double) * lda * m , hipMemcpyHostToDevice); 
	cudaStat2 = hipMemcpy(d_B, B, sizeof(double) * ldb * nrhs, hipMemcpyHostToDevice); 
	assert(hipSuccess == cudaStat1); 
	assert(hipSuccess == cudaStat2);

	//Call the solver (starts here)
	
	// step 3: query working space of geqrf and ormqr
	cusolver_status = hipsolverDnDgeqrf_bufferSize( 
		cusolverH, 
		m, 
		m, 
		d_A, 
		lda, 
		&lwork); 
	assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
	
	cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);
	
	// step 4: compute QR factorization
	cusolver_status = hipsolverDnDgeqrf( 
		cusolverH, 
		m, 
		m, 
		d_A, 
		lda, 
		d_tau, 
		d_work, 
		lwork, 
		devInfo);
	cudaStat1 = hipDeviceSynchronize(); 
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status); 
	assert(hipSuccess == cudaStat1); 
	
	// check if QR is good or not 
	cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost); 
	assert(hipSuccess == cudaStat1); 
	
	printf("after geqrf: info_gpu = %d\n", info_gpu); 
	assert(0 == info_gpu);
	
	// step 5: compute Q^T*B
	cusolver_status= hipsolverDnDormqr( 
		cusolverH, 
		HIPBLAS_SIDE_LEFT, 
		HIPBLAS_OP_T, 
		m, 
		nrhs, 
		m, 
		d_A, 
		lda, 
		d_tau, 
		d_B, 
		ldb, 
		d_work, 
		lwork, 
		devInfo);
	cudaStat1 = hipDeviceSynchronize(); 
	assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status); 
	assert(hipSuccess == cudaStat1);
	
	//Check the results (starts here)
	
	// check if QR is good or not
	cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost); 
	assert(hipSuccess == cudaStat1); 
	
	printf("after ormqr: info_gpu = %d\n", info_gpu); 
	assert(0 == info_gpu); 
	
	// step 6: compute x = R \ Q^T*B 
	
	cublas_status = hipblasDtrsm( 
		cublasH, 
		HIPBLAS_SIDE_LEFT, 
		HIPBLAS_FILL_MODE_UPPER, 
		HIPBLAS_OP_N, 
		HIPBLAS_DIAG_NON_UNIT, 
		m, 
		nrhs, 
		&one, 
		d_A, 
		lda, 
		d_B, 
		ldb); 
	cudaStat1 = hipDeviceSynchronize(); 
	assert(HIPBLAS_STATUS_SUCCESS == cublas_status); 
	assert(hipSuccess == cudaStat1); 
	
	//copy result back to the host from device and print it
	cudaStat1 = hipMemcpy(XC, d_B, sizeof(double)*ldb*nrhs, hipMemcpyDeviceToHost); 
	assert(hipSuccess == cudaStat1); 
	
	//printf("X = (matlab base-1)\n"); 
	//printMatrix(m, nrhs, XC, ldb, "X");
	
	// free resources
	if (d_A ) hipFree(d_A); 
	if (d_tau ) hipFree(d_tau); 
	if (d_B ) hipFree(d_B); 
	if (devInfo) hipFree(devInfo); 
	if (d_work ) hipFree(d_work); 
	
	if (cublasH ) hipblasDestroy(cublasH); 
	if (cusolverH) hipsolverDnDestroy(cusolverH); 
	
	hipDeviceReset();
}

int main()
{
	
	const int H_A_m = 3; 
	const int H_A_n = H_A_m;
	const int H_B_m = H_A_n; 
	const int H_B_n = 1;
	
	//Create the library handle and load the data (starts here)
	
	double A[H_A_m*H_A_n] = { 21.81678168,  15.31087255,  26.18776594, 15.31087255,  13.93152484,  23.92113599, 26.18776594,  23.92113599,  41.50060023};
	
	double B[H_B_m*H_B_n] = { 15.94772944,  16.57202022,  28.49909096}; 
	const int H_XC_m = H_B_m;
	const int H_XC_n = H_B_n;
	//double *B; 
	
	double *XC;  // solution matrix from GPU 
	
	XC = (double *)malloc(H_XC_m*H_XC_n);
	
	lin_alg_solve(XC, A, B, H_A_m, H_B_n);
	
	printMatrix(H_A_m, H_B_n, XC, H_A_n, "X");
	
}


